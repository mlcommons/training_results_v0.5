#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/extension.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCNumerics.cuh>
#include <THC/THC.h>
#include <hip/hip_runtime.h>
#include <vector>

__device__
float4 add_boxes(const float4& a, const float4& b) {
  return float4{a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
}

/**
 * Essentially bolis down to a grid-stride loop
 * - get the index of the output and backtrack what its values should
 *   be based on the arange that would have been created.
 * - Easy parallelism - use (BSZ_X, A) as block dimensions to parallelize over
 *   the A anchors used.
 * - Accesses to global memory are all done via. float4
 */
__global__
void generate_anchors_single(const int image_height,
                             const int image_width,
                             const int feature_height,
                             const int feature_width,
                             const float4* anchor_data, // [1, 3, 4]
                             const int stride,
                             const int A,
                             float4 *anchors,
                             const float straddle_thresh,
                             uint8_t* inds_inside) {

  // size of arange is floor(start - end / step)
  // in this case, floor((feature{height,width} * stride - 0) / stride)
  const int len_x = (int)floorf(feature_width);
  const int len_y = (int)floorf(feature_height);

#if 0
  if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0) {
    printf("len_x: %d, len_y: %d\n", len_x, len_y);
  }
#endif
  // Standard grid-stride loop over output size
  for (int output_idx = threadIdx.x + blockIdx.x * blockDim.x;
           output_idx < len_x * len_y;
           output_idx += gridDim.x * blockDim.x) {
    // local box is (xp, yp, xp, yp)
    // where xp = x[output_idx % len(x)]
    //       yp = y[output_idx / len(y)]
    // and x = (output_idx % len(x)) * step
    //     y = (output_idx / len(y)) * step
    const float x = (output_idx % len_x) * stride;
    const float y = (output_idx / len_x) * stride;

    // This is the basic box
    float4 box{x, y, x, y};

    // parallelize over anchors
    const int i = threadIdx.y;
    // for (int i = 0; i < A; ++i) {
    const float4 a = anchor_data[i];

    float4 tmp = add_boxes(box, a);

    anchors[output_idx * A + i] = tmp;

    // for each anchor, now check
    if (straddle_thresh >= 0.f) {
      inds_inside[output_idx * A + i] = (tmp.x >= -straddle_thresh)
                                      & (tmp.y >= -straddle_thresh)
                                      & (tmp.z < image_width + straddle_thresh)
                                      & (tmp.w < image_height + straddle_thresh);
    } else {
      inds_inside[output_idx * A + i] = 1;
    }
  }
}


std::vector<at::Tensor> anchor_generator(
    std::vector<int64_t> image_shape,       // (height, width)
    std::vector<int64_t> feature_map_size,  // (height, width)
    at::Tensor& cell_anchors,               // shape: [1, 3, 4]
    const int stride,
    const float straddle_thresh) {

  // Need to work out some sizes for the kernel
  const float h_start = 0.;
  const float h_end = feature_map_size[0] * stride;
  const int h_elems = (int)std::floor( (h_end - h_start) / stride );

  const float w_start = 0., w_end = feature_map_size[1] * stride;
  const int w_elems = (int)std::floor( (w_end - w_start) / stride );

  // If cell anchors are [A, 4]
  const int A = cell_anchors.size(0);
  // output anchors are h_elems * w_elems * A * 4 values, so allocate that now.
  at::Tensor anchors = at::zeros({h_elems * w_elems * A, 4}, torch::CUDA(at::kFloat));
  // also output a bool map of anchors being inside the image
  at::Tensor inds_inside = at::zeros({h_elems * w_elems * A}, torch::CUDA(at::kByte));

  // CUDA grid is going to be (32, A) * (h_elems * w_elems / 32)
  const int blockx = 64;
  dim3 block(blockx, A);
  dim3 grid((h_elems * w_elems + (blockx - 1)) / blockx);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  generate_anchors_single<<<grid, block, 0, stream>>>(
                             image_shape[0],
                             image_shape[1],
                             feature_map_size[0],
                             feature_map_size[1],
                             reinterpret_cast<float4*>(cell_anchors.data<float>()),
                             stride,
                             A,
                             reinterpret_cast<float4*>(anchors.data<float>()),
                             straddle_thresh,
                             inds_inside.data<uint8_t>());
  THCudaCheck(hipGetLastError());

  return {anchors, inds_inside};
}



